// ---------------------------------------------------------------------
//
// Copyright (C) 2019 by the deal.II authors
//
// This file is part of the deal.II library.
//
// The deal.II library is free software; you can use it, redistribute
// it, and/or modify it under the terms of the GNU Lesser General
// Public License as published by the Free Software Foundation; either
// version 2.1 of the License, or (at your option) any later version.
// The full text of the license can be found in the file LICENSE.md at
// the top level directory of deal.II.
//
// ---------------------------------------------------------------------


#include <hip/hip_runtime.h>
#include <iostream>

int main()
{
  hipDeviceProp_t device_properties;
  const hipError_t error = hipGetDeviceProperties(&device_properties,
                                                    /*device*/0);
  if( error != hipSuccess)
  {
    std::cout << "CUDA error: " << hipGetErrorString(error) << '\n';
    return error;
  }
  std::cout << device_properties.major << device_properties.minor;
  return 0;
}
